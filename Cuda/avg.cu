#include "hip/hip_runtime.h"

#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include <string.h>
// Kernel function to add the elements of two arrays
__global__
void sum_cuda(int iter, int n, float* x)
{
    uint64_t pick = 1 << iter;
    uint64_t index = (pick*2)*(blockIdx.x * blockDim.x + threadIdx.x);
    uint64_t stride = (pick*2)*blockDim.x * gridDim.x;
    for(uint64_t i = index; i + pick < n ; i += stride){
        x[i] = x[i] + x[i+pick];
    }
    // x[index] = x[index] + x[index+pick];

}

float avg_cuda(int n, float* x){
    int block_size = 256;
    for(int i = 0; n > (1 << i); i++){
        int num_blocks = max((n + block_size - 1)/max((2*block_size*(1 << i)), 1), 1);
        sum_cuda<<<num_blocks, block_size>>>(i, n, x);
    }
    hipDeviceSynchronize();
    return *x/n;
}

float avg_cpu(int n, float* x){
    double total = 0;
    for(int i = 0; i < n; i++){
        total += x[i];
    }
    return total/n;
}

int main(void)
{
    int N = 1<<6+1;
    float *x;
    float *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    // srand(clock());
    for (int i = 0; i < N; i++) {
        x[i] = (float)(rand()%100);
        y[i] = x[i];
        // x[i] = (float)(i%2);
    }

    int time, ptime = clock();
    float true_avg = avg_cpu(N, y);
    time = clock();
    printf("cpu time elapsed: %f sec\n", ((float)(time-ptime))/CLOCKS_PER_SEC);

    memcpy(y, x, N*sizeof(float));
    ptime = clock();
    float calculated_avg = avg_cuda(N, y);
    time = clock();
    printf("cuda time elapsed: %f sec\n", ((float)(time-ptime))/CLOCKS_PER_SEC);

    printf("cuda: %f  \t cpu (expected): %f\n", calculated_avg, true_avg);
    // for(int i = 0; i < 20; i++){
    //     printf("%f\n", x[i]);
    // }

    // Free memory
    hipFree(x);

    return 0;
}