#include "hip/hip_runtime.h"


#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include <string.h>

inline int ceil_log2(unsigned long long x)
{
  static const unsigned long long t[6] = {
    0xFFFFFFFF00000000ull,
    0x00000000FFFF0000ull,
    0x000000000000FF00ull,
    0x00000000000000F0ull,
    0x000000000000000Cull,
    0x0000000000000002ull
  };

  int y = (((x & (x - 1)) == 0) ? 0 : 1);
  int j = 32;
  int i;

  for (i = 0; i < 6; i++) {
    int k = (((x & t[i]) == 0) ? 0 : j);
    y += k;
    x >>= k;
    j >>= 1;
  }

  return y;
}

// Kernel function to avg the elements of array
__global__
void sum_cuda(int iter, int n, float* x)
{
    uint64_t pick = 1 << iter;
    uint64_t index = (pick*2)*(blockIdx.x * blockDim.x + threadIdx.x);
    uint64_t stride = (pick*2)*blockDim.x * gridDim.x;
    for(uint64_t i = index; i + pick < n ; i += stride){
        x[i] = x[i] + x[i+pick];
    }
    // x[index] = x[index] + x[index+pick];
}

__global__
void sum_cuda2(uint32_t stride, uint32_t n, float* x)
{
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    for(uint64_t i = index; i + stride < n; i += blockDim.x * gridDim.x){
        x[i] = x[i] + x[i+stride];
    }
}

float avg_cuda(int n, float* x){
    int block_size = 128;
    for(int i = 0; n > (1 << i); i++){
        int num_blocks = max((n + block_size - 1)/max((2*block_size*(1 << i)), 1), 1);
        sum_cuda<<<num_blocks, block_size>>>(i, n, x);
    }
    hipDeviceSynchronize();
    return *x/n;
}

float avg_cuda2(int n, float* x){
    int block_size = 128;
    for(int i = 0; n > (1 << i); i++){
        int num_blocks = min(max((n+1)/2, 1), 32);
        uint32_t stride = 1 << (ceil_log2(n) - i - 1);
        sum_cuda2<<<num_blocks, block_size>>>(stride, n, x);
    }
    hipDeviceSynchronize();
    return *x/n;
}

float avg_cpu(int n, float* x){
    double total = 0;
    for(int i = 0; i < n; i++){
        total += x[i];
    }
    return total/n;
}

int main(void)
{
    int N = 1<<20;
    float *x;
    float *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    // srand(clock());
    for (int i = 0; i < N; i++) {
        x[i] = (float)(rand()%100);
        // x[i] = (float)(i%2);
        y[i] = x[i];
    }

    int time, ptime = clock();
    float true_avg = avg_cpu(N, y);
    time = clock();
    printf("cpu time elapsed:  \t%f sec\n", ((float)(time-ptime))/CLOCKS_PER_SEC);

    memcpy(y, x, N*sizeof(float));
    ptime = clock();
    float calculated_avg = avg_cuda2(N, y);
    time = clock();
    printf("cuda time elapsed: \t%f sec\n", ((float)(time-ptime))/CLOCKS_PER_SEC);

    printf("cuda: %f  \t cpu (expected): %f\n", calculated_avg, true_avg);
    // for(int i = 0; i < 20; i++){
    //     printf("%f\n", x[i]);
    // }

    // Free memory
    hipFree(x);

    return 0;
}