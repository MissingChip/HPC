
#include <hip/hip_runtime.h>

void sum_cpu(int n, int iter,float *x)
{
    int pick = 1 << iter;
    int index = pick;
    int stride = (pick*2);
    printf("iter %d\n", iter);
    for(int i = index; i + pick < n ; i += stride){
        printf("%d: %f %f %f\n", i, x[i] + x[i+pick], x[i], x[pick]);
        x[i] = x[i] + x[i+pick];
    }
}


float avg_emul(int n, float* x){
    for(int i = 0; n > (1 << i); i++){
        sum_cpu(n, i, x);
    }
    return *x/n;
}


int ceil_log2(unsigned long long x)
{
  static const unsigned long long t[6] = {
    0xFFFFFFFF00000000ull,
    0x00000000FFFF0000ull,
    0x000000000000FF00ull,
    0x00000000000000F0ull,
    0x000000000000000Cull,
    0x0000000000000002ull
  };

  int y = (((x & (x - 1)) == 0) ? 0 : 1);
  int j = 32;
  int i;

  for (i = 0; i < 6; i++) {
    int k = (((x & t[i]) == 0) ? 0 : j);
    y += k;
    x >>= k;
    j >>= 1;
  }

  return y;
}

__global__
void sum_cuda2(int iter, int n, float* x)
{
    uint64_t pick = pow(4, iter);
    uint64_t index = (pick*4)*(blockIdx.x * blockDim.x + threadIdx.x);
    uint64_t stride = (pick*4)*blockDim.x * gridDim.x;
    for(uint64_t i = index; i + pick*3 < n ; i += stride){
        x[i] = x[i] + x[i+pick] + x[i+pick*2] + x[i+pick*3];
    }
}
